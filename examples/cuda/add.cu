/*
 * Eduardo Berrocal
 * eberroca@iit.edu
 *
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#define CHECK_ERR(x)                                    \
  if (x != hipSuccess) {                               \
    fprintf(stderr,"%s in %s at line %d\n",             \
        hipGetErrorString(err),__FILE__,__LINE__);	\
    exit(-1);											\
  }                                                     \
  
__global__ void vecAdd (float* d_A, float* d_B, float* d_C, int n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n)
    d_C[i] = d_A[i] + d_B[i];
}

int main() {

  hipError_t err;

  int n = 10;
  // Memory allocation for h_A, h_B and h_C (in the host)
  float h_A[10] = { 1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0, 8.0, 9.0, 0.0 };
  float h_B[10] = { 0.0, 9.0, 8.0, 7.0, 6.0, 5.0, 4.0, 3.0, 2.0, 1.0 };
  float h_C[10];

  float *d_A, *d_B, *d_C;

  // Memory allocation for d_A, d_B and d_C (in the device)
  err = hipMalloc((void **) &d_A, sizeof(float)*n);
  CHECK_ERR(err);

  err =hipMalloc((void **) &d_B, sizeof(float)*n);
  CHECK_ERR(err);

  err =hipMalloc((void **) &d_C, sizeof(float)*n);
  CHECK_ERR(err);

  // Copying memory to device
  err = hipMemcpy(d_A, h_A, sizeof(float)*n, hipMemcpyHostToDevice);
  CHECK_ERR(err);

  err = hipMemcpy(d_B, h_B, sizeof(float)*n, hipMemcpyHostToDevice);
  CHECK_ERR(err);


  // Calling the kernel
  vecAdd<<<ceil(n/256.0), 256>>>(d_A,d_B,d_C,n);

  // Copying results back to host
  err = hipMemcpy(h_C, d_C, sizeof(float)*n, hipMemcpyDeviceToHost);

  int i = 0;
  for (; i < 10; i++)
    printf("h_C[%d] = %f\n",i,h_C[i]);

}